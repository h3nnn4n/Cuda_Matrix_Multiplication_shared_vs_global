#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>

#include "info.h"
#include "err.h"

__global__ void matrix_mul(int *a, int *b, int *c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    c[row * N + col] = 0;

    for ( int i = 0; i  < N ; i ++ ) {
        c[row * N + col] += a[row * N + i] * b[i * N + col];
    }
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N2 * sizeof(int);

    struct timeval timevalA;
    struct timeval timevalB;

    struct timeval timevalA2;
    struct timeval timevalB2;

    gettimeofday(&timevalA2,NULL);

    gpuErrchk( hipMalloc( (void**) &d_a, size));
    gpuErrchk( hipMalloc( (void**) &d_b, size));
    gpuErrchk( hipMalloc( (void**) &d_c, size));

    a = (int*) malloc ( size );
    b = (int*) malloc ( size );
    c = (int*) malloc ( size );

    if ( a == NULL ) { fprintf(stderr, "Failed to allocate a\n"); abort(); }
    if ( b == NULL ) { fprintf(stderr, "Failed to allocate b\n"); abort(); }
    if ( c == NULL ) { fprintf(stderr, "Failed to allocate c\n"); abort(); }

    for ( int i = 0; i  < N ; i ++ ) {
        for ( int j = 0; j  < N ; j ++ ) {
            a[i*N + j] = 0;
            b[i*N + j] = 0;
            c[i*N + j] = 0;
            if ( i == j ) {
                a[i*N + j] = 1;
                b[i*N + j] = 1;
                c[i*N + j] = 1;
            }
        }
    }

#ifdef __output
    printf("Matrix size = %d %d\n", N, N);
    printf("Number of elements = %d\n", N2);
    printf("Grid size = %d %d \n", NBLOCKS, NBLOCKS);
    printf("Number of grid elements = %d\n", NBLOCKS * NBLOCKS);
    printf("Number of elements per grid = %d\n", THREADS_PER_BLOCK * THREADS_PER_BLOCK);
    printf("Threads per block = %d %d\n", THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    printf("Check %d = %d\n", THREADS_PER_BLOCK2 * NBLOCKS * NBLOCKS, N2);
#endif

    gpuErrchk( hipMemcpy( d_a, a, size, hipMemcpyHostToDevice ));
    gpuErrchk( hipMemcpy( d_b, b, size, hipMemcpyHostToDevice ));

    dim3 block  = dim3(NBLOCKS,
                       NBLOCKS,
                       1                  );
    dim3 thread = dim3(THREADS_PER_BLOCK,
                       THREADS_PER_BLOCK,
                       1                 );

    gettimeofday(&timevalA,NULL);
    matrix_mul<<< block, thread >>>(d_a, d_b, d_c);
    gettimeofday(&timevalB,NULL);


    gpuErrchk( hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost ));

    gettimeofday(&timevalB2,NULL);

    /*for ( int i = 0; i  < N ; i ++ ) {*/
        /*for ( int j = 0; j  < N ; j ++ ) {*/
            /*printf("%d ", c[i*N + j]);*/
        /*}*/
        /*printf("\n");*/
    /*}*/

    printf("%d %f %f\n", N, timevalB.tv_sec-timevalA.tv_sec+(timevalB.tv_usec-timevalA.tv_usec)/(double)1000000,
                            timevalB2.tv_sec-timevalA2.tv_sec+(timevalB2.tv_usec-timevalA2.tv_usec)/(double)1000000
          );

    free(a);
    free(b);
    free(c);
    gpuErrchk( hipFree(a));
    gpuErrchk( hipFree(b));
    gpuErrchk( hipFree(c));

    return 0;
}
