#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>

#include "info.h"
#include "err.h"

__global__ void matrix_mul(int *a, int *b, int *c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int posx = threadIdx.x;
    int posy = threadIdx.y;

    int step = 0;

    for ( int w = 0; w < N/SHARED_BLOCK_SIZE; w++ ) {

        __shared__ int as[SHARED_BLOCK_SIZE * SHARED_BLOCK_SIZE];
        __shared__ int bs[SHARED_BLOCK_SIZE * SHARED_BLOCK_SIZE];

        as[posy * SHARED_BLOCK_SIZE + posx] = a[blockIdx.y * blockDim.x * N + w * blockDim.x     + posx + posy * N ];
        bs[posy * SHARED_BLOCK_SIZE + posx] = b[blockIdx.x * blockDim.x     + w * blockDim.x * N + posx + posy * N ];

        __syncthreads();

        for ( int i = 0; i  < SHARED_BLOCK_SIZE ; i ++ ) {
            step += as[posy * SHARED_BLOCK_SIZE + i   ] *
                    bs[i    * SHARED_BLOCK_SIZE + posx];
        }

        __syncthreads();
    }

    c[row * N + col] = step;
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N2 * sizeof(int);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    gpuErrchk( hipMalloc( (void**) &d_a, size));
    gpuErrchk( hipMalloc( (void**) &d_b, size));
    gpuErrchk( hipMalloc( (void**) &d_c, size));

    a = (int*) malloc ( size );
    b = (int*) malloc ( size );
    c = (int*) malloc ( size );

    if ( a == NULL ) { fprintf(stderr, "Failed to allocate a\n"); abort(); }
    if ( b == NULL ) { fprintf(stderr, "Failed to allocate b\n"); abort(); }
    if ( c == NULL ) { fprintf(stderr, "Failed to allocate c\n"); abort(); }

    for ( int i = 0; i  < N ; i ++ ) {
        for ( int j = 0; j  < N ; j ++ ) {
            a[i*N + j] = 0;
            b[i*N + j] = 0;
            c[i*N + j] = 0;
            if ( i == j ) {
                a[i*N + j] = 1;
                b[i*N + j] = 1;
                c[i*N + j] = 1;
            }
        }
    }

#ifdef __output
    printf("Matrix size = %d %d\n", N, N);
    printf("Number of elements = %d\n", N2);
    printf("Grid size = %d %d \n", NBLOCKS, NBLOCKS);
    printf("Shared size = %d %d \n", SHARED_BLOCK_SIZE, SHARED_BLOCK_SIZE);
    printf("Number of grid elements = %d\n", NBLOCKS * NBLOCKS);
    printf("Number of elements per grid = %d\n", THREADS_PER_BLOCK * THREADS_PER_BLOCK);
    printf("Threads per block = %d %d\n", THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    printf("Check %d = %d\n", THREADS_PER_BLOCK2 * NBLOCKS * NBLOCKS, N2);
#endif

    gpuErrchk( hipMemcpy( d_a, a, size, hipMemcpyHostToDevice ));
    gpuErrchk( hipMemcpy( d_b, b, size, hipMemcpyHostToDevice ));

    dim3 block  = dim3(NBLOCKS,
                       NBLOCKS,
                       1                  );
    dim3 thread = dim3(THREADS_PER_BLOCK,
                       THREADS_PER_BLOCK,
                       1                 );

    hipEventRecord(start);
    matrix_mul<<< block, thread >>>(d_a, d_b, d_c);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    /*CudaCheckError();*/

    gpuErrchk( hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost ));

    for ( int i = 0; i  < N ; i ++ ) {
        for ( int j = 0; j  < N ; j ++ ) {
            if ( c[i*N + j] != 0  && i != j ) {
                fprintf(stderr, "Found nonzero outside the main diagonal\n");
                abort();
            } else if ( c[i*N + j] != 1  && i == j ) {
                fprintf(stderr, "Found something not 1 in the main diagonal\n");
                abort();
            }
            /*printf("%d", c[i*N + j]);*/
        }
        /*printf("\n");*/
    }
    /*printf("Matrix ok\n");*/

    printf("%d %f\n", N, milliseconds);

    free(a);
    free(b);
    free(c);
    gpuErrchk( hipFree(d_a));
    gpuErrchk( hipFree(d_b));
    gpuErrchk( hipFree(d_c));

    return 0;
}
